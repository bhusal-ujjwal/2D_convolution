#include "hip/hip_runtime.h"
#include <iostream>
#include <png.h>
#include "utils/pngio.h"
#include "hip/hip_runtime.h"

#define CUDA_CHECK_RETURN(value) \
    { hipError_t err = value; \
      if (err != hipSuccess) { \
        fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(err), __LINE__, __FILE__); \
        exit(1); \
      } }

#define FILTER_SIZE 3
#define BLOCK_SIZE 16
#define TAIL_SIZE (BLOCK_SIZE - (FILTER_SIZE / 2) * 2)
#define SHARED_MEM_SIZE ((BLOCK_SIZE + FILTER_SIZE - 1) * (BLOCK_SIZE + FILTER_SIZE - 1))

__constant__ float CONVOLUTION_MASKS[3][FILTER_SIZE * FILTER_SIZE] = {
    {-1, -1, -1, -1, 8, -1, -1, -1, -1},             // Ridge
    {1.0/9, 1.0/9, 1.0/9, 1.0/9, 1.0/9, 1.0/9, 1.0/9, 1.0/9, 1.0/9}, // Normalized Box Blur
    {0, -1, 0, -1, 5, -1, 0, -1, 0}                   // Sharpen
};

__global__ void applyConvolution(unsigned char *output, const unsigned char *input, int width, int height, int maskIndex) {
    __shared__ unsigned char sharedMem[BLOCK_SIZE + FILTER_SIZE - 1][BLOCK_SIZE + FILTER_SIZE - 1];

    int x = threadIdx.x + blockIdx.x * TAIL_SIZE;
    int y = threadIdx.y + blockIdx.y * TAIL_SIZE;
    int sharedX = threadIdx.x + FILTER_SIZE / 2;
    int sharedY = threadIdx.y + FILTER_SIZE / 2;

    if (x < width && y < height) {
        int offset = FILTER_SIZE / 2;
        float result[3] = {0.0f, 0.0f, 0.0f};

        for (int c = 0; c < 3; ++c) {
            for (int i = -offset; i <= offset; ++i)
                for (int j = -offset; j <= offset; ++j)
                    sharedMem[sharedY + j][sharedX + i] = input[(((y + j) * width) + (x + i)) * 3 + c];

            __syncthreads();

            for (int i = -offset; i <= offset; ++i)
                for (int j = -offset; j <= offset; ++j)
                    result[c] += sharedMem[sharedY + j][sharedX + i] * CONVOLUTION_MASKS[maskIndex][j + offset + (i + offset) * FILTER_SIZE];

            __syncthreads();
        }

        if (x < width && y < height) {
            for (int c = 0; c < 3; ++c) {
                output[((y * width + x) * 3) + c] = static_cast<unsigned char>(result[c]);
            }
        }
    }
}

int main() {
    png::image<png::rgb_pixel> img("../lenna.png");
    unsigned int width = img.get_width();
    unsigned int height = img.get_height();
    unsigned int size = width * height * 3 * sizeof(unsigned char);

    unsigned char *h_data = new unsigned char[size];
    pvg::pngToRgb(h_data, img);

    unsigned char *d_input, *d_output;

    CUDA_CHECK_RETURN(hipMalloc(&d_input, size));
    CUDA_CHECK_RETURN(hipMalloc(&d_output, size));

    hipMemcpy(d_input, h_data, size, hipMemcpyHostToDevice);

    dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_dim((width + TAIL_SIZE - 1) / TAIL_SIZE, (height + TAIL_SIZE - 1) / TAIL_SIZE);

    for (int i = 0; i < 3; ++i) {
        applyConvolution<<<grid_dim, block_dim>>>(d_output, d_input, width, height, i);
        hipDeviceSynchronize();
        CUDA_CHECK_RETURN(hipGetLastError());

        hipMemcpy(h_data, d_output, size, hipMemcpyDeviceToHost);
        pvg::rgbToPng(img, h_data);
        img.write("../lenna_new_" + std::to_string(i) + ".png");
    }

    hipFree(d_input);
    hipFree(d_output);
    delete[] h_data;

    return 0;
}
